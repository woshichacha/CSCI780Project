
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>

timeval StartingTime;

void setTime(){
	gettimeofday( &StartingTime, NULL );
}

double getTime(){
	timeval PausingTime, ElapsedTime;
	gettimeofday( &PausingTime, NULL );
	timersub(&PausingTime, &StartingTime, &ElapsedTime);
	return ElapsedTime.tv_sec*1000.0+ElapsedTime.tv_usec/1000.0;	// Returning in milliseconds.
}
