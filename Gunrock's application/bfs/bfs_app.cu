// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file bfs_app.cu
 *
 * @brief Gunrock breadth-first search (BFS) application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// breadth-first search includes
#include <gunrock/app/bfs/bfs_enactor.cuh>
#include <gunrock/app/bfs/bfs_problem.cuh>
#include <gunrock/app/bfs/bfs_functor.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bfs;

/**
 * @brief BFS_Parameter structure
 */
struct BFS_Parameter : gunrock::app::TestParameter_Base
{
public:
    bool   mark_predecessors ;  // mark src-distance vs. parent vertices
    bool   enable_idempotence;  // enable idempotence operation
    bool   direction_optimized; // enable direction optimization
    double max_queue_sizing1 ;  // maximum queue sizing factor

    BFS_Parameter()
    {
        mark_predecessors  = false;
        enable_idempotence = false;
        direction_optimized = false;
        max_queue_sizing1  = -1.0f;
    }

    ~BFS_Parameter()
    {
    }
};

template<typename VertexId, typename SizeT, typename Value,
         //bool INSTRUMENT, bool DEBUG, bool SIZE_CHECK,
         bool MARK_PREDECESSORS, bool ENABLE_IDEMPOTENCE>
float runBFS(GRGraph* output, BFS_Parameter *parameter);

/**
 * @brief Run test
 *
 * @tparam VertexId          Vertex identifier type
 * @tparam Value             Attribute type
 * @tparam SizeT             Graph size type
 * @tparam MARK_PREDECESSORS Enable mark predecessors
 *
 * @param[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 *
 * \return Elapsed run time in milliseconds
 */
template <
    typename    VertexId,
    typename    SizeT,
    typename    Value,
    bool        MARK_PREDECESSORS >
float RunTests_enable_idempotence(GRGraph* output, BFS_Parameter *parameter)
{
    if (parameter->enable_idempotence)
        return runBFS<VertexId, SizeT, Value,/*INSTRUMENT, DEBUG,
               SIZE_CHECK,*/ MARK_PREDECESSORS,  true>(output, parameter);
    else
        return runBFS<VertexId, SizeT, Value,/*INSTRUMENT, DEBUG,
               SIZE_CHECK,*/ MARK_PREDECESSORS, false>(output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 *
 * @param[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 *
 * \return Elapsed run time in milliseconds
 */
template <
    typename    VertexId,
    typename    SizeT,
    typename    Value>
float RunTests_mark_predecessors(GRGraph* output, BFS_Parameter *parameter)
{
    if (parameter->mark_predecessors)
        return RunTests_enable_idempotence
            <VertexId, SizeT, Value, true>(output, parameter);
    else
        return RunTests_enable_idempotence
            <VertexId, SizeT, Value, false>(output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId           Vertex identifier type
 * @tparam Value              Attribute type
 * @tparam SizeT              Graph size type
 * @tparam MARK_PREDECESSORS  Enable mark predecessors
 * @tparam ENABLE_IDEMPOTENCE Enable idempotent operation
 *
 * @param[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 *
 * \return Elapsed run time in milliseconds
 */
template <
    typename    VertexId,
    typename    SizeT,
    typename    Value,
    bool        MARK_PREDECESSORS,
    bool        ENABLE_IDEMPOTENCE >
float runBFS(GRGraph* output, BFS_Parameter *parameter)
{
    typedef BFSProblem < VertexId,
            SizeT,
            Value,
            MARK_PREDECESSORS,
            ENABLE_IDEMPOTENCE>
            //(MARK_PREDECESSORS && ENABLE_IDEMPOTENCE) >
            Problem; // does not use double buffer

    typedef BFSEnactor < Problem>
            //INSTRUMENT,
            //DEBUG,
            //SIZE_CHECK >
            Enactor;

    Csr<VertexId, SizeT, Value> *graph =
        (Csr<VertexId, SizeT, Value>*)parameter->graph;
    bool          quiet                = parameter -> g_quiet;
    int           max_grid_size        = parameter -> max_grid_size;
    int           num_gpus             = parameter -> num_gpus;
    int           num_iters            = parameter -> iterations;
    double        max_queue_sizing     = parameter -> max_queue_sizing;
    double        max_queue_sizing1    = parameter -> max_queue_sizing1;
    double        max_in_sizing        = parameter -> max_in_sizing;
    ContextPtr   *context              = (ContextPtr*)parameter -> context;
    std::string   partition_method     = parameter -> partition_method;
    int          *gpu_idx              = parameter -> gpu_idx;
    hipStream_t *streams              = parameter -> streams;
    float         partition_factor     = parameter -> partition_factor;
    int           partition_seed       = parameter -> partition_seed;
    bool          g_stream_from_host   = parameter -> g_stream_from_host;
    std::string   traversal_mode       = parameter -> traversal_mode;
    bool          instrument           = parameter -> instrumented;
    bool          debug                = parameter -> debug;
    bool          size_check           = parameter -> size_check;
    bool          undirected           = parameter -> g_undirected;
    bool          direction_optimized  = parameter -> direction_optimized;
    size_t       *org_size             = new size_t  [num_gpus];
    // Allocate host-side label array
    VertexId     *h_labels             = new VertexId[graph->nodes];
    VertexId     *h_preds              = NULL;

    if (MARK_PREDECESSORS)
    {
        h_preds = new VertexId[graph->nodes];
    }

    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }
    Problem *problem = new Problem(direction_optimized, undirected);  // Allocate problem on GPU

    util::GRError( problem->Init(
        g_stream_from_host,
        graph,
        NULL,
        num_gpus,
        gpu_idx,
        partition_method,
        streams,
        max_queue_sizing,
        max_in_sizing,
        partition_factor,
        partition_seed),
        "Problem BFS Initialization Failed", __FILE__, __LINE__);

    Enactor *enactor = new Enactor(
        num_gpus, gpu_idx, instrument, debug, size_check, direction_optimized);  // BFS enactor map
    util::GRError(
        enactor->Init(context, problem, max_grid_size, traversal_mode),
        "BFS Enactor init failed", __FILE__, __LINE__);

    CpuTimer cpu_timer;
    float elapsed = 0.0f;
    for (int i = 0; i < num_iters; ++i)
    {
        printf("Round %d of bfs.\n", i+1);
        util::GRError(
                problem->Reset(parameter->src[i], enactor->GetFrontierType(),
                    max_queue_sizing, max_queue_sizing1),
                "BFS Problem Data Reset Failed", __FILE__, __LINE__);
        util::GRError(
                enactor->Reset(), "BFS Enactor Reset failed", __FILE__, __LINE__);

        cpu_timer.Start();

        util::GRError(
                enactor->Enact(parameter->src[i], traversal_mode),
                "BFS Problem Enact Failed", __FILE__, __LINE__);

        cpu_timer.Stop();

        elapsed += cpu_timer.ElapsedMillis();
    }

    // Copy out results
    util::GRError(
        problem->Extract(h_labels, h_preds),
        "BFS Problem Data Extraction Failed", __FILE__, __LINE__);

    output->node_value1 = (Value*)&h_labels[0];
    if (MARK_PREDECESSORS) output->node_value2 = (VertexId*)&h_preds[0];

    if (!quiet)
    {
        printf(" GPU Breath-First Search finished in %lf msec.\n", elapsed);
    }

    // Clean up
    if (org_size) delete[] org_size; org_size = NULL;
    if (enactor ) delete   enactor ; enactor  = NULL;
    if (problem ) delete   problem ; problem  = NULL;

    return elapsed;
}

/**
 * @brief Dispatch function to handle configurations
 *
 * @param[out] grapho  Pointer to output graph structure of the problem
 * @param[in]  graphi  Pointer to input graph we need to process on
 * @param[in]  config  Primitive-specific configurations
 * @param[in]  data_t  Data type configurations
 * @param[in]  context ModernGPU context
 * @param[in]  streams CUDA stream
 *
 * \return Elapsed run time in milliseconds
 */
float dispatch_bfs(
    GRGraph*       grapho,
    const GRGraph* graphi,
    const GRSetup* config,
    const GRTypes  data_t,
    ContextPtr*    context,
    hipStream_t*  streams)
{
    BFS_Parameter *parameter = new BFS_Parameter;
    parameter->iterations = config -> num_iters;
    parameter->src = (long long*)malloc(sizeof(long long)*config -> num_iters);
    parameter->context  = context;
    parameter->streams  = streams;
    parameter->g_quiet  = config -> quiet;
    parameter->num_gpus = config -> num_devices;
    parameter->gpu_idx  = config -> device_list;
    parameter->mark_predecessors  = config -> mark_predecessors;
    parameter->enable_idempotence = config -> enable_idempotence;

    float elapsed_time;

    switch (data_t.VTXID_TYPE)
    {
    case VTXID_INT:
    {
        switch (data_t.SIZET_TYPE)
        {
        case SIZET_INT:
        {
            switch (data_t.VALUE_TYPE)
            {
            case VALUE_INT:    // template type = <int, int, int>
            {
                // build input CSR format graph
                Csr<int, int, int> csr(false);
                csr.nodes = graphi->num_nodes;
                csr.edges = graphi->num_edges;
                csr.row_offsets    = (int*)graphi->row_offsets;
                csr.column_indices = (int*)graphi->col_indices;
                parameter->graph = &csr;

                // determine source vertex to start
                switch (config -> source_mode)
                {
                case randomize:
                {
                    for (int i = 0; i < parameter->iterations; ++i)
                    {
                        parameter->src[i] = graphio::RandomNode(csr.nodes);
                    }
                    break;
                }
                case largest_degree:
                {
                    int max_deg = 0;
                    int node_id = csr.GetNodeWithHighestDegree(max_deg);
                    for (int i = 0; i < config -> num_iters; ++i)
                    {
                        parameter->src[i] = node_id;
                    }
                    break;
                }
                case manually:
                {
                    for (int i = 0; i < parameter->iterations; ++i)
                    {
                        parameter->src[i] = config -> source_vertex[i];
                    }
                    break;
                }
                default:
                {
                    for (int i = 0; i < parameter->iterations; ++i)
                    {
                        parameter->src[i] = 0;
                    }
                    break;
                }
                }
                if (!parameter->g_quiet)
                {
                    printf(" source: %lld", (long long) parameter->src[0]);
                    for (int i = 1; i < config -> num_iters; ++i)
                    {
                        printf(",%lld", (long long) parameter->src[i]);
                    }
                    printf("\n");
                }

                elapsed_time = RunTests_mark_predecessors<int, int, int>(grapho, parameter);

                // reset for free memory
                csr.row_offsets    = NULL;
                csr.column_indices = NULL;
                break;
            }
            case VALUE_UINT:    // template type = <int, uint, int>
            {
                // not yet support
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_FLOAT:    // template type = <int, float, int>
            {
                // not yet support
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            }
            break;
        }
        }
        break;
    }
    }
    free(parameter->src);
    return elapsed_time;
}

/*
 * @brief Entry of gunrock_bfs function
 *
 * @param[out] grapho Pointer to output graph structure of the problem
 * @param[in]  graphi Pointer to input graph we need to process on
 * @param[in]  config Gunrock primitive specific configurations
 * @param[in]  data_t Gunrock data type structure
 */
float gunrock_bfs(
    GRGraph*       grapho,
    const GRGraph* graphi,
    const GRSetup* config,
    const GRTypes  data_t)
{
    // GPU-related configurations
    int           num_gpus =    0;
    int           *gpu_idx = NULL;
    ContextPtr    *context = NULL;
    hipStream_t  *streams = NULL;

    num_gpus = config -> num_devices;
    gpu_idx  = new int [num_gpus];
    for (int i = 0; i < num_gpus; ++i)
    {
        gpu_idx[i] = config -> device_list[i];
    }

    // Create streams and MordernGPU context for each GPU
    streams = new hipStream_t[num_gpus * num_gpus * 2];
    context = new ContextPtr[num_gpus * num_gpus];
    if (!config -> quiet) { printf(" using %d GPUs:", num_gpus); }
    for (int gpu = 0; gpu < num_gpus; ++gpu)
    {
        if (!config -> quiet) { printf(" %d ", gpu_idx[gpu]); }
        util::SetDevice(gpu_idx[gpu]);
        for (int i = 0; i < num_gpus * 2; ++i)
        {
            int _i = gpu * num_gpus * 2 + i;
            util::GRError(hipStreamCreate(&streams[_i]),
                          "hipStreamCreate fialed.", __FILE__, __LINE__);
            if (i < num_gpus)
            {
                context[gpu * num_gpus + i] =
                    mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],
                                                       streams[_i]);
            }
        }
    }
    if (!config -> quiet) { printf("\n"); }

    return dispatch_bfs(grapho, graphi, config, data_t, context, streams);
}

/*
 * @brief Simple interface take in CSR arrays as input
 *
 * @param[out] bfs_label            Return BFS label (depth) per nodes or the predecessor per nodes
 * @param[in]  num_nodes            Number of nodes of the input graph
 * @param[in]  num_edges            Number of edges of the input graph
 * @param[in]  row_offsets          CSR-formatted graph input row offsets
 * @param[in]  col_indices          CSR-formatted graph input column indices
 * @param[in]  num_iters            Number of BFS runs. Note if num_iters > 1, the bfs_lbel will only store the results from the last run
 * @param[in]  source               Sources to begin traverse
 * @param[in]  source_mode          Enumerator of source mode: manually, randomize, largest_degree
 * @param[in]  mark_predecessors    If the flag is set, mark predecessors instead of bfs label
 * @param[in]  enable_idempotence   If the flag is set, use optimizations that allow idempotence operation (will usually bring better performance)
 */
float bfs(
    int*       bfs_label,
    int*       bfs_preds,
    const int  num_nodes,
    const int  num_edges,
    const int* row_offsets,
    const int* col_indices,
    const int  num_iters,
    int* source,
    enum SrcMode source_mode,
    const bool mark_predecessors,
    const bool enable_idempotence)
{
    struct GRTypes data_t;          // primitive-specific data types
    data_t.VTXID_TYPE = VTXID_INT;  // integer vertex identifier
    data_t.SIZET_TYPE = SIZET_INT;  // integer graph size type
    data_t.VALUE_TYPE = VALUE_INT;  // integer attributes type

    struct GRSetup* config = InitSetup(num_iters, source);  // primitive-specific configures
    config -> mark_predecessors  = mark_predecessors;    // do not mark predecessors
    config -> enable_idempotence = enable_idempotence;    // wether enable idempotence
    config -> source_mode = source_mode;

    struct GRGraph *grapho = (struct GRGraph*)malloc(sizeof(struct GRGraph));
    struct GRGraph *graphi = (struct GRGraph*)malloc(sizeof(struct GRGraph));

    graphi->num_nodes   = num_nodes;  // setting graph nodes
    graphi->num_edges   = num_edges;  // setting graph edges
    graphi->row_offsets = (void*)&row_offsets[0];  // setting row_offsets
    graphi->col_indices = (void*)&col_indices[0];  // setting col_indices


    float elapsed_time = gunrock_bfs(grapho, graphi, config, data_t);
    memcpy(bfs_label, (int*)grapho->node_value1, num_nodes * sizeof(int));
    if (mark_predecessors) 
        memcpy(bfs_preds, (int*)grapho->node_value2, num_nodes * sizeof(int));

    if (graphi) free(graphi);
    if (grapho) free(grapho);
    if (config) free(config);

    return elapsed_time;
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
