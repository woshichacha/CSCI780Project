// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file pr_app.cu
 *
 * @brief Gunrock PageRank application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// page-rank includes
#include <gunrock/app/pr/pr_enactor.cuh>
#include <gunrock/app/pr/pr_problem.cuh>
#include <gunrock/app/pr/pr_functor.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::pr;

/**
 * @brief PR_Parameter structure
 */
struct PR_Parameter : gunrock::app::TestParameter_Base
{
public:
    float    delta          ;  // Delta value for PageRank
    float    error          ;  // Error threshold PageRank
    int      max_iter       ;  // Maximum number of iteration
    bool     normalized     ;

    PR_Parameter()
    {
        delta    = 0.85f;
        error    = 0.01f;
        max_iter =    50;
        normalized = false;
    }

    ~PR_Parameter()
    {
    }
};

template <
    typename VertexId,
    typename SizeT,
    typename Value,
    //bool INSTRUMENT,
    //bool DEBUG,
    //bool SIZE_CHECK,
    bool NORMALIZED>
void runPageRank(GRGraph *output, PR_Parameter *parameter);

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 * @tparam DEBUG      Keep debug statics
 *
 * @param[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename      VertexId,
    typename      SizeT,
    typename      Value>
    //bool          INSTRUMENT,
    //bool          DEBUG,
    //bool          SIZE_CHECK >
void normalizedPageRank(GRGraph *output, PR_Parameter *parameter)
{
    if (parameter -> normalized)
        runPageRank<VertexId, SizeT, Value, true > (output, parameter);
    else
        runPageRank<VertexId, SizeT, Value, false> (output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 * @tparam DEBUG      Keep debug statics
 * @tparam SIZE_CHECK Enable size check
 *
 * @param[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <
    typename VertexId,
    typename SizeT,
    typename Value,
    //bool INSTRUMENT,
    //bool DEBUG,
    //bool SIZE_CHECK,
    bool NORMALIZED >
void runPageRank(GRGraph *output, PR_Parameter *parameter)
{
    typedef PRProblem < VertexId,
            SizeT,
            Value,
            NORMALIZED > Problem;

    typedef PREnactor < Problem>
            //INSTRUMENT,
            //DEBUG,
            //SIZE_CHECK > 
            Enactor;

    Csr<VertexId, SizeT, Value>
    *graph              = (Csr<VertexId, SizeT, Value>*)parameter->graph;
    bool          quiet              = parameter -> g_quiet;
    int           max_grid_size      = parameter -> max_grid_size;
    int           num_gpus           = parameter -> num_gpus;
    double        max_queue_sizing   = parameter -> max_queue_sizing;
    double        max_in_sizing      = parameter -> max_in_sizing;
    ContextPtr   *context            = (ContextPtr*)parameter -> context;
    std::string   partition_method   = parameter -> partition_method;
    int          *gpu_idx            = parameter -> gpu_idx;
    hipStream_t *streams            = parameter -> streams;
    float         partition_factor   = parameter -> partition_factor;
    int           partition_seed     = parameter -> partition_seed;
    bool          g_stream_from_host = parameter -> g_stream_from_host;
    VertexId      src                = parameter -> src[0];
    Value         delta              = parameter -> delta;
    Value         error              = parameter -> error;
    SizeT         max_iter           = parameter -> max_iter;
    std::string   traversal_mode     = parameter -> traversal_mode;
    bool          instrument         = parameter -> instrumented;
    bool          debug              = parameter -> debug;
    bool          size_check         = parameter -> size_check;
    size_t       *org_size           = new size_t  [num_gpus];
    // Allocate host-side label arrays
    Value        *h_rank             = new Value   [graph->nodes];
    VertexId     *h_node_id          = new VertexId[graph->nodes];

    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }

    Problem *problem = new Problem(false);  // Allocate problem on GPU
    util::GRError(
        problem->Init(
            g_stream_from_host,
            graph,
            NULL,
            num_gpus,
            gpu_idx,
            partition_method,
            streams,
            context,
            max_queue_sizing,
            max_in_sizing,
            partition_factor,
            partition_seed),
        "PR Initialization Failed", __FILE__, __LINE__);

    Enactor* enactor = new Enactor(
        num_gpus, gpu_idx, instrument, debug, size_check);  // enactor map
    util::GRError(
        enactor->Init(context, problem, traversal_mode, max_grid_size),
        "PR Enactor Init failed", __FILE__, __LINE__);

    // Perform PageRank
    CpuTimer cpu_timer;

    util::GRError(
        problem->Reset(src, delta, error, max_iter,
                       enactor->GetFrontierType(), max_queue_sizing),
        "PR Problem Data Reset Failed", __FILE__, __LINE__);
    util::GRError(
        enactor->Reset(), "PR Enactor Reset Reset failed", __FILE__, __LINE__);

    cpu_timer.Start();
    util::GRError(
        enactor->Enact(traversal_mode),
        "PR Problem Enact Failed", __FILE__, __LINE__);
    cpu_timer.Stop();

    float elapsed = cpu_timer.ElapsedMillis();

    // Copy out results
    util::GRError(
        problem->Extract(h_rank, h_node_id),
        "PR Problem Data Extraction Failed", __FILE__, __LINE__);

    double total_pr = 0;
    for (SizeT i = 0; i < graph->nodes; ++i)
    {
        total_pr += h_rank[i];
    }
    if (!quiet) { printf(" Total rank : %.8e\n", total_pr); }

    output->node_value1 = (Value*)&h_rank[0];
    output->node_value2 = (VertexId*)&h_node_id[0];

    if (!quiet) { printf(" GPU PageRank finished in %lf msec.\n", elapsed); }

    // Clean up
    if (org_size) { delete[] org_size; org_size = NULL; }
    if (problem ) { delete problem ; problem  = NULL; }
    if (enactor ) { delete enactor ; enactor  = NULL; }
}

/**
 * @brief Dispatch function to handle configurations
 *
 * @param[out] grapho  Pointer to output graph structure of the problem
 * @param[in]  graphi  Pointer to input graph we need to process on
 * @param[in]  config  Primitive-specific configurations
 * @param[in]  data_t  Data type configurations
 * @param[in]  context ModernGPU context
 * @param[in]  streams CUDA stream
 */
void dispatchPageRank(
    GRGraph       *grapho,
    const GRGraph *graphi,
    const GRSetup *config,
    const GRTypes  data_t,
    ContextPtr*    context,
    hipStream_t*  streams)
{
    PR_Parameter *parameter = new PR_Parameter;
    parameter->src = (long long*)malloc(sizeof(long long));
    parameter->src[0] = -1;
    parameter->context      =  context;
    parameter->streams      =  streams;
    parameter->g_quiet      = config -> quiet;
    parameter->num_gpus     = config -> num_devices;
    parameter->gpu_idx      = config -> device_list;
    parameter->delta        = config -> pagerank_delta;
    parameter->error        = config -> pagerank_error;
    parameter->max_iter     = config -> max_iters;
    parameter->normalized   = config -> pagerank_normalized;
    parameter->g_undirected = true;

    switch (data_t.VTXID_TYPE)
    {
    case VTXID_INT:
    {
        switch (data_t.SIZET_TYPE)
        {
        case SIZET_INT:
        {
            switch (data_t.VALUE_TYPE)
            {
            case VALUE_INT:    // template type = <int, int, int>
            {
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_UINT:    // template type = <int, uint, int>
            {
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_FLOAT:    // template type = <int, float, int>
            {
                // build input csr format graph
                Csr<int, int, int> csr(false);
                csr.nodes = graphi->num_nodes;
                csr.edges = graphi->num_edges;
                csr.row_offsets    = (int*)graphi->row_offsets;
                csr.column_indices = (int*)graphi->col_indices;
                parameter->graph = &csr;

                normalizedPageRank<int, int, float>(grapho, parameter);

                // reset for free memory
                csr.row_offsets    = NULL;
                csr.column_indices = NULL;
                break;
            }
            }
            break;
        }
        }
        break;
    }
    }
    free(parameter->src);
}

/*
 * @brief Entry of gunrock_pagerank function
 *
 * @param[out] grapho Pointer to output graph structure of the problem
 * @param[in]  graphi Pointer to input graph we need to process on
 * @param[in]  config Gunrock primitive specific configurations
 * @param[in]  data_t Gunrock data type structure
 */
void gunrock_pagerank(
    GRGraph       *grapho,
    const GRGraph *graphi,
    const GRSetup *config,
    const GRTypes  data_t)
{
    // GPU-related configurations
    int           num_gpus =    0;
    int           *gpu_idx = NULL;
    ContextPtr    *context = NULL;
    hipStream_t  *streams = NULL;

    num_gpus = config -> num_devices;
    gpu_idx  = new int [num_gpus];
    for (int i = 0; i < num_gpus; ++i)
    {
        gpu_idx[i] = config -> device_list[i];
    }

    // Create streams and MordernGPU context for each GPU
    streams = new hipStream_t[num_gpus * num_gpus * 2];
    context = new ContextPtr[num_gpus * num_gpus];
    if (!config -> quiet) { printf(" using %d GPUs:", num_gpus); }
    for (int gpu = 0; gpu < num_gpus; ++gpu)
    {
        if (!config -> quiet) { printf(" %d ", gpu_idx[gpu]); }
        util::SetDevice(gpu_idx[gpu]);
        for (int i = 0; i < num_gpus * 2; ++i)
        {
            int _i = gpu * num_gpus * 2 + i;
            util::GRError(hipStreamCreate(&streams[_i]),
                          "hipStreamCreate fialed.", __FILE__, __LINE__);
            if (i < num_gpus)
            {
                context[gpu * num_gpus + i] =
                    mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],
                                                       streams[_i]);
            }
        }
    }
    if (!config -> quiet) { printf("\n"); }

    dispatchPageRank(grapho, graphi, config, data_t, context, streams);
}

/*
 * @brief Simple interface take in CSR arrays as input
 *
 * @param[out] node_ids    Return top-ranked vertex IDs
 * @param[out] pagerank    Return PageRank scores per node
 * @param[in]  num_nodes   Number of nodes of the input graph
 * @param[in]  num_edges   Number of edges of the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 * @param[in]  source      Source to begin traverse
 */
void pagerank(
    int*                node_ids,
    float*              pagerank,
    const int           num_nodes,
    const int           num_edges,
    const int*          row_offsets,
    const int*          col_indices,
    bool                normalized)
{
    struct GRTypes data_t;            // primitive-specific data types
    data_t.VTXID_TYPE = VTXID_INT;    // integer vertex identifier
    data_t.SIZET_TYPE = SIZET_INT;    // integer graph size type
    data_t.VALUE_TYPE = VALUE_FLOAT;  // float attributes type

    struct GRSetup *config = InitSetup(1, NULL);  // primitive-specific configures
    config -> top_nodes      = 10;           // number of top nodes
    config -> pagerank_normalized     = normalized;

    struct GRGraph *grapho = (struct GRGraph*)malloc(sizeof(struct GRGraph));
    struct GRGraph *graphi = (struct GRGraph*)malloc(sizeof(struct GRGraph));

    graphi->num_nodes   = num_nodes;  // setting graph nodes
    graphi->num_edges   = num_edges;  // setting graph edges
    graphi->row_offsets = (void*)&row_offsets[0];  // setting row_offsets
    graphi->col_indices = (void*)&col_indices[0];  // setting col_indices

    gunrock_pagerank(grapho, graphi, config, data_t);
    memcpy(pagerank, (float*)grapho->node_value1, num_nodes * sizeof(float));
    memcpy(node_ids, (  int*)grapho->node_value2, num_nodes * sizeof(  int));

    if (graphi) free(graphi);
    if (grapho) free(grapho);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
